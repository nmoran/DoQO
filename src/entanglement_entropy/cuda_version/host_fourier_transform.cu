#include "hip/hip_runtime.h"
#include "host_fourier_transform.h"

#define PI 3.141592653589793f

// calculated the momentum that each boson is at and populates the monomials array with this. Does so in ascending order.
void get_monomial_bosons(unsigned long Index, int Sites, int Bosons, int *Monomial) 
  {
    int mom = 1;
    int idx = 0;
    int pos = 0;
    while ( pos < (Sites + Bosons) ) {
      if ( (Index & (1ul << pos) ) > 0ul ) {
	  Monomial[idx++] = mom;
      } else {
	  mom += 1;
      }	  
      pos++;
    }  
  }

// calculated the momentum that each boson is at and populates the monomials array with this. Does so in ascending order and also return the normal.
void get_monomial_bosons(unsigned long Index, int Sites, int Bosons, int *Monomial, float *Normal) 
  {
    int mom = 1;
    int idx = 0;
    int pos = 0;
    *Normal = 1.0;
    double num = 0;
    while ( pos < (Sites + Bosons) ) {
      if ( (Index & (1ul << pos) ) > 0ul ) {
	  Monomial[idx++] = mom;
	  num += 1.0;
	  *Normal *= num;
      } else {
	  mom += 1;
	  //if ( num > 1.0 ) *normal *= num; 
	  num = 0.0; 
      }	  
      pos++;
    }
    //if ( num > 1.0 ) *normal *= num; 
    *Normal = sqrt(*Normal);
  }


// calculated the momentum that each boson is at and populates the monomials array with this. Does so in ascending order.
void get_monomial_fermions(unsigned long Index, int Sites, int Fermions, int *Monomial) 
  {
    int mom = 1;
    int idx = 0;
    int pos = 0;
    while ( pos < Sites ) {
      if ( (Index & (1ul << pos) ) > 0ul ) {
	  Monomial[idx++] = mom;
	  mom += 1; 
      } else {
	  mom += 1;
      }	  
      pos++;
    }  
  }

int permute(int *Monomial, int Len)
  {
    int k,l,tmp;
    
    k = Len - 2;
    while (k > 0 && Monomial[k] >= Monomial[k+1] ) k--;
    
    if ( k == 0 && (Monomial[0] >= Monomial[1]) ) return 0;
    
    l = Len - 1;
    while ( Monomial[l] <= Monomial[k] ) l--;
    
    tmp = Monomial[k];
    Monomial[k] = Monomial[l];
    Monomial[l] = tmp;
    
    l = Len - 1;
    k = k + 1;
    
    while ( k < l ) 
    {
      tmp = Monomial[k];
      Monomial[k] = Monomial[l];
      Monomial[l] = tmp;
      l--;
      k++;
    }
    return 1;       
  }

float configuration_normal(unsigned long Config, int NumberSites, unsigned long Mask)
  {
    float normal = 1.0;    
    
    for ( int i = 1 ; i < NumberSites ; i++ ) 
    {
      if ( Config == ( ((Config << i) | (Config >> (NumberSites - i))) & Mask ) ) 
	{
	  normal += 1.0;
	}
    }
    return 1.0/sqrt(normal);             
  }

void fourier_transform_vector(int VectorLength, float *VectorElements, unsigned long *BasisElements, int FTBasisDim, float *FTVectorElements, unsigned long *FTBasisElements, int NumberSites, int Filling, int CurrentMomentum, int SpinMomentum)
  {
    int *fmonomial, *fmonomialTmp, *omonomial;
    fmonomial = new int[Filling];
    fmonomialTmp = new int[Filling];
    omonomial = new int[Filling];
    
    float *phaseArrayR = new float[NumberSites+1];
    float *phaseArrayI = new float[NumberSites+1];
    phaseArrayR[0] = 1.0;
    phaseArrayI[0] = 0.0;
    for( int i = 1  ; i <= NumberSites ; i++ ) 
      {	    	    				
        double angle = ( PI * 2.0 * (double)i) / ((double)NumberSites);
	phaseArrayR[i] = cos(angle);
	phaseArrayI[i] = sin(angle);
	//cout << "Phase array[" << i << "] = " << phaseArrayR[i] << endl;
      }
    
    float norm = sqrt(1.0/pow((double)NumberSites,(double)Filling));
    
    float momentumPhase = 1.0;
    if ( SpinMomentum != -1 ) 
      {
	for ( int i = 1; i < NumberSites ; i++ )
	  {
	    momentumPhase += phaseArrayR[(i * (CurrentMomentum + SpinMomentum)) % NumberSites];
	  }
      }
    norm *= sqrt(momentumPhase);
    
    int phaseIdx;
    float tmp_val, tmp_val2;
    float stateNormal = 1.0;
    
    if ( SpinMomentum == -1 ) 
      {
	for ( int fidx = 0 ; fidx < FTBasisDim ; fidx++ )
	  {
	    tmp_val = 0.0;
	    get_monomial_bosons(FTBasisElements[fidx], NumberSites, Filling, fmonomial, &stateNormal);
	    get_monomial_bosons(FTBasisElements[fidx], NumberSites, Filling, fmonomialTmp, &stateNormal);
	    for ( int oidx = 0 ; oidx < VectorLength ; oidx++ ) 
	      {
		//sort(fmonomial, fmonomial + Filling); //reset to original configuration.
		memcpy(fmonomial, fmonomialTmp, sizeof(int)*Filling);
		tmp_val2 = 0.0;
		get_monomial_fermions(BasisElements[oidx], NumberSites, Filling, omonomial);
		do 
		  {
		    phaseIdx = 0;
		    for ( int i = 0 ; i < Filling; i++ )
		      {
			phaseIdx += omonomial[i] * fmonomial[i];	            
		      }
		    tmp_val2 += phaseArrayR[phaseIdx % NumberSites];
		  }
		while ( permute(fmonomial, Filling) );
		tmp_val += tmp_val2 * VectorElements[oidx];
	      }	
	    FTVectorElements[fidx] = tmp_val * norm * stateNormal;
	  }
      }
    else 
      {
	unsigned long mask = (1ul << NumberSites) - 1ul;
	for ( int fidx = 0 ; fidx < FTBasisDim ; fidx++ )
	  {
	    tmp_val = 0.0;
	    get_monomial_bosons(FTBasisElements[fidx], NumberSites, Filling, fmonomial, &stateNormal);
	    get_monomial_bosons(FTBasisElements[fidx], NumberSites, Filling, fmonomialTmp, &stateNormal);
	    for ( int oidx = 0 ; oidx < VectorLength ; oidx++ ) 
	      {
		//sort(fmonomial, fmonomial + Filling); //reset to original configuration.
		memcpy(fmonomial, fmonomialTmp, sizeof(int)*Filling);
		tmp_val2 = 0.0;
		get_monomial_fermions(BasisElements[oidx], NumberSites, Filling, omonomial);
		do 
		  {
		    phaseIdx = 0;
		    for ( int i = 0 ; i < Filling; i++ )
		      {
			phaseIdx += omonomial[i] * fmonomial[i];	            
		      }
		    tmp_val2 += phaseArrayR[phaseIdx % NumberSites];
		  }
		while ( permute(fmonomial, Filling) );
		tmp_val += tmp_val2 * VectorElements[oidx] * configuration_normal(BasisElements[oidx], NumberSites, mask);
	      }	
	    FTVectorElements[fidx] = tmp_val * norm * stateNormal;
	  }
      }
    delete [] fmonomial;
    delete [] fmonomialTmp;
    delete [] omonomial;
    delete [] phaseArrayR;
    delete [] phaseArrayI;
  }
