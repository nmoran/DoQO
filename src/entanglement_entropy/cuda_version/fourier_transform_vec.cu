#include "hip/hip_runtime.h"
#include "host_fourier_transform.h"
#include "device_fourier_transform.h"


/*
 This function calculates the dimension of the boson basis set with given total momentum.
 
 sites : number of sites.
 filling : number of bosons.
 M : total momentum.
*/
int boson_basis_dimension(int sites, int filling, int M)
{  
  if ( sites > 0 && sites == M && filling == 1 )
    {
      return 1;
    }  
    
  if ( M < filling || M > sites*filling || sites == 0 || filling == 0 ) 
    {
      return 0; 
    } 
  else 
    {
    
      int dim = 0;  
      //use recursive procedure where can add boson and stay in position, add boson and move on or not add boson and move on.
      if ( sites <= M ) 
	{
	  dim += boson_basis_dimension(sites, filling-1, M-sites);	  
	}
      dim += boson_basis_dimension(sites-1, filling, M);
      return dim; 
    }
}


/*
 This function calculates the dimension of the boson basis set with given total momentum.
 
 sites : number of sites.
 filling : number of bosons.
 M : total momentum.
 states: array of states.
 maxm : the maximum m for a given state.
*/
int boson_basis_generate(int sites, int filling, int M, int pos, unsigned long *states, unsigned long  current_state)
{  
  if ( sites > 0 && sites == M && filling == 1 )
    {      
      current_state += 1ul << (sites + filling - 2);      
      states[pos++] = current_state;      
      return pos;
    }  
    
  if ( M < filling || M > sites*filling || sites == 0 || filling == 0 ) 
    {
      return pos; 
    } 
  else 
    {          
      //use recursive procedure where can add boson and stay in position, add boson and move on or not add boson and move on.
      pos = boson_basis_generate(sites-1, filling, M, pos, states, current_state);
      if ( sites <= M ) 
	{
	  current_state += 1ul << (sites + filling - 2);	  
	  pos = boson_basis_generate(sites, filling-1, M-sites, pos, states, current_state);
	}
      
      return pos; 
    }
}


int main ( int argc, char **argv) 
{
    int c;
    string inputFilename, basisFilename, outputPrefix; 
    int numberSites = 0, filling = 0, momentum = -1, spinMomentum = -1;
    bool truncate = false, useCuda = false, single = false, benchmark = false, benchmark2 = false;
    int rank, size;
    int cuda_device = 0;
    int benchmarkSize = 0;
    int benchmark2Size = 0;
    int offset = 0;
    
    MPI_Init (&argc, &argv);	/* starts MPI */
    MPI_Comm_rank (MPI_COMM_WORLD, &rank);	/* get current process id */
    MPI_Comm_size (MPI_COMM_WORLD, &size);	/* get number of processes */    
     
    //Read command line arguments.
    while (1)
      {
	static struct option long_options[] =
	  {	    
	    {"vec",  required_argument,  0, 'v'},
	    {"basis",  required_argument,  0, 'b'},
	    {"num_sites",  required_argument,  0, 'n'},
	    {"filling",  required_argument,  0, 'f'},
	    {"truncate",  no_argument,  0, 't'},
	    {"momentum",  required_argument,  0, 'm'},
	    {"spin_momentum",  required_argument,  0, 'k'},
	    {"device", required_argument, 0, 'd'},
	    {"cuda",  no_argument,  0, 'c'},
	    {"single", no_argument, 0, 's'},
	    {"benchmark", required_argument, 0, 'x'},
	    {"benchmark2", required_argument, 0, 'y'},
	    {"offset", required_argument, 0, 'o'},
	    {0, 0, 0, 0}
	  };
	/* getopt_long stores the option index here. */
	int option_index = 0;
  
	c = getopt_long (argc, argv, "v:b:n:f:tm:k:csd:x:y:o:",
			long_options, &option_index);
  
	/* Detect the end of the options. */
	if (c == -1)
	  break;
  
	switch (c)
	  {	  
	    case 'v':
	      inputFilename.assign(optarg);
	      size_t found;
	      found = inputFilename.find_last_of(".");
	      outputPrefix.assign(inputFilename.substr(0,found));
	      break;  	  
	    case 'b':
	      basisFilename.assign(optarg);	      
	      break;
	    case 'n':
	      numberSites = atoi(optarg);
	      break;
	    case 'f':
	      filling = atoi(optarg);
	      break;
	    case 't':
	      truncate = true;
	      break;
	    case 'm':
	      momentum = atoi(optarg);
	      break;
	    case 'k':
	      spinMomentum = atoi(optarg);
	      break;
	    case 'c':
	      useCuda = true;
	      break;
	    case 's':
	      single = true;
	      break;
	    case 'd':
	      cuda_device = atoi(optarg);
	      break;
	    case 'x':
	      benchmark = true;
	      benchmarkSize = atoi(optarg);
	      break;  
	    case 'y':
	      benchmark2 = true;
	      benchmark2Size = atoi(optarg);
	      break;  
	    case 'o':
	      offset = atoi(optarg);
	      break;  
  
	    default:
	      MPI_Finalize();
	      abort ();
	  }
      }
      
    //Check required arguments were supplied.
    if ( inputFilename == "" ) 
      {
	cout << "No input filename supplied." << endl;
	return 1;
      }
    if ( basisFilename == "" ) 
      {
	cout << "No basis filename supplied." << endl;
	return 1;
      }
    if ( numberSites  <= 0 ) 
      {
	cout << "Invalid number of sites." << endl;
	return 1;
      }
    if ( filling  <= 0 ) 
      {
	cout << "Invalid filling." << endl;
	return 1;
      }
      
    //Print out values of arguements read.
    cout << "Input filename: " << inputFilename << endl;
    cout << "Basis filename: " << basisFilename << endl;
    cout << "Number sites: " << numberSites << endl;
    cout << "Filling: " << filling << endl;
    cout << "Truncation: " << truncate << endl;
    cout << "CUDA enabled: " << useCuda << endl;
    
    //Now read in the vector and basis. 
    //Will read in vector as just real floats for now and basis will be long integers.
    int vectorLength = 0;
    float *vectorElements;
    ifstream vectorFile(inputFilename.c_str());
    if ( vectorFile.is_open() ) 
      {
	  string line;
	  getline(vectorFile,line);
	  while ( line != ""  ) 
	    {
	      vectorLength++;
	      getline(vectorFile,line);
	    }
	  vectorFile.close();
	  vectorElements = new float[vectorLength];
	  vectorLength = 0;
	  vectorFile.open(inputFilename.c_str());
	  if ( vectorFile.is_open() ) 
	    {
		string line;
		getline(vectorFile,line);
		while ( line != ""  ) 
		  {		    		    
		    vectorElements[vectorLength++] = atof(line.c_str());
		    getline(vectorFile,line);
		  }
		vectorFile.close();
	    }
	}
    else
      {
	cout << "Problem opening " << inputFilename << endl;
	return 1;
      }
      
    int basisLength = 0;
    unsigned long *basisElements;
    ifstream basisFile(basisFilename.c_str());
    if ( basisFile.is_open() ) 
      {
	  string line;
	  getline(basisFile,line); //skip a line for the size part
	  getline(basisFile,line);
	  while ( line != "" ) 
	    {
	      basisLength++;
	      getline(basisFile,line);
	    }
	  basisFile.close();
	  basisElements = new unsigned long[basisLength];
	  basisLength = 0;
	  basisFile.open(basisFilename.c_str());
	  if ( basisFile.is_open() ) 
	    {
		string line;
		getline(basisFile,line); //skip a line for the size part
		getline(basisFile,line);
		while ( line != ""  ) 
		  {		    		    
		    basisElements[basisLength++] = atol(line.c_str());
		    getline(basisFile,line);
		  }
		basisFile.close();
	    }
	}
    else
      {
	cout << "Problem opening " << inputFilename << endl;
	return 1;
      }
      
    if ( vectorLength != basisLength ) 
      {
	cout << "Vector and basis lengths do not match." << endl;
	return 1;
      }
    
    cout << "Input vector size: " << vectorLength << endl;
      
    // cout << "Vector and basis details." << endl;
//     for ( int i = 0 ; i < vectorLength ; i++ ) 
//       {
// 	cout << basisElements[i] << "\t: " << vectorElements[i] << endl;
//       }
      
    int startMomentum = 0; 
    int endMomentum = numberSites * filling;
    if ( momentum >= 0 && momentum <= (numberSites * filling) ) 
      {
	startMomentum = momentum;
	endMomentum = momentum;
      }
      
    //if we are using translational invariance then apply normal now.  
    if ( spinMomentum != -1 ) 
      {
	unsigned long mask = (1ul << numberSites) - 1ul;
	for ( int i = 0 ; i < vectorLength ; i++ ) 
	  {
	    vectorElements[i] *= configuration_normal(basisElements[i], numberSites, mask);
	  }  
      }
    
    for ( int currentMomentum = startMomentum ; currentMomentum <= endMomentum ; currentMomentum++ )
      {
	//Now need to calculate the basis that the fourier transformed vector will use.
	int FTBasisDim = boson_basis_dimension(numberSites, filling, currentMomentum);
	unsigned long * FTBasisElements = new unsigned long [FTBasisDim];
	float* FTVectorElements = new float[FTBasisDim];
	for ( int i = 0 ; i < FTBasisDim ; i++ ) 
	  {
	    FTVectorElements[i] = 0.0;
	  }
	boson_basis_generate(numberSites, filling, currentMomentum, 0, FTBasisElements, 0ul);
	cout << "Sector with total momentum " << currentMomentum << " has " << FTBasisDim << " elements." << endl;
		
	if ( benchmark ) 
	  {
	    if ( benchmarkSize > 0 ) 
	      {
		unsigned long *FTBasisBenchmarkElements = new unsigned long [benchmarkSize];
		int bidx = 0, fidx = 0;
		float normal;
		int monomial[filling];
		while ( (bidx < benchmarkSize) && (fidx < FTBasisDim) )
		  {
		    get_monomial_bosons(FTBasisElements[fidx], numberSites, filling, monomial, &normal);
		    if ( normal == 1.0 ) 
		      {
			FTBasisBenchmarkElements[bidx++] = FTBasisElements[fidx];
		      }
		    fidx++;
		  }	
		if ( bidx < benchmarkSize ) 
		  {
		    cout << "Only using " << bidx << " elements for benchmark out of requested " << benchmarkSize << endl;
		    benchmarkSize = bidx; 
		  }
		if ( useCuda ) 
		  {
		    double start, finish;	   
		    start = MPI_Wtime();
		    if ( benchmark2 && benchmark2Size <= vectorLength ) 
		      {
			fourier_transform_vector_cuda(benchmark2Size, vectorElements, basisElements, benchmarkSize, FTVectorElements, FTBasisBenchmarkElements, numberSites, filling, currentMomentum, spinMomentum, single, cuda_device, outputPrefix, offset);
		      }
		    else
		      {
			fourier_transform_vector_cuda(vectorLength, vectorElements, basisElements, benchmarkSize, FTVectorElements, FTBasisBenchmarkElements, numberSites, filling, currentMomentum, spinMomentum, single, cuda_device, outputPrefix, offset);	
		      }
		    finish = MPI_Wtime();  		
		    cout << "Fourier transform of " << benchmarkSize << " benchmark elements using CUDA for momentum " << currentMomentum << " took " << finish - start << " seconds." << endl;
		  } 
		else 
		  {
		    double start, finish;	   
		    start = MPI_Wtime();
		    if ( benchmark2 && benchmark2Size <= vectorLength ) 
		      {
			fourier_transform_vector(benchmark2Size, vectorElements, basisElements, benchmarkSize, FTVectorElements, FTBasisBenchmarkElements, numberSites, filling, currentMomentum, spinMomentum);
		      }
		    else
		      {
			fourier_transform_vector(vectorLength, vectorElements, basisElements, benchmarkSize, FTVectorElements, FTBasisBenchmarkElements, numberSites, filling, currentMomentum, spinMomentum);	
		      }
		    finish = MPI_Wtime();  		
		    cout << "Fourier transform of " << benchmarkSize << " benchmark elements for momentum " << currentMomentum << " took " << finish - start << " seconds." << endl;
		  }
		delete []FTBasisBenchmarkElements;
	      }	    
	  }
	else 
	  {
	    if ( useCuda ) 
	      {
		double start, finish;	   
		start = MPI_Wtime();
		fourier_transform_vector_cuda(vectorLength, vectorElements, basisElements, FTBasisDim, FTVectorElements, FTBasisElements, numberSites, filling, currentMomentum, spinMomentum, single, cuda_device, outputPrefix, offset);	
		finish = MPI_Wtime();
		cout << "Fourier transform using CUDA for momentum " << currentMomentum << " took " << finish - start << " seconds." << endl;
	      }
	    else
	      {				
		double start, finish;
		start = MPI_Wtime();
		fourier_transform_vector(vectorLength, vectorElements, basisElements, FTBasisDim, FTVectorElements, FTBasisElements, numberSites, filling, currentMomentum, spinMomentum);	
		finish = MPI_Wtime();
		cout << "Fourier transform for momentum " << currentMomentum << " took " << finish - start << " seconds." << endl;
		
	      }
	  }	      	      	
	
	float mag = 0;
	for( int i = 0 ; i < FTBasisDim ; i++ )
	  {
	    mag += FTVectorElements[i] * FTVectorElements[i];
	  }
	cout.precision(14);
	cout << scientific << "FT vector magnitude: " << sqrt(mag) << endl;
// 	for( int i = 0 ; i < FTBasisDim ; i++ )
// 	  {
// 	    cout.precision(14);
// 	    cout << scientific << "Element " << i << ": " << FTVectorElements[i] << endl;
// 	  }  	

	//output basis and vector elements. 
	string outputFilename;
	stringstream ss("");
	ss << outputPrefix << "_FT_M_" << currentMomentum << ".vec" ; 
	ofstream out(ss.str().c_str());
	if ( out.is_open() )
	  {
	    out.precision(14);
	    for ( int i = FTBasisDim - 1 ; i >= 0 ; i-- ) 
	      {
		out << scientific << FTVectorElements[i] << endl;
	      }
	    out.close();
	  }	  
	ss.str("");
	ss << outputPrefix << "_FT_M_" << currentMomentum << ".basis" ; 
	out.open(ss.str().c_str());
	if ( out.is_open() )
	  {	    
	    for ( int i = FTBasisDim - 1 ; i >= 0 ; i-- ) 
	      {
		out << FTBasisElements[i] << endl;
	      }
	    out.close();
	  }	  	
	delete [] FTBasisElements;
	delete [] FTVectorElements;
      }  
      
    delete [] basisElements;
    delete [] vectorElements;
    MPI_Finalize();
  
    return 0;
}