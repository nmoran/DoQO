#include "hip/hip_runtime.h"
#include "device_fourier_transform.h"
#include "host_fourier_transform.h"
 
#define CUDA_BLOCKS 510  		//this should be a multiple of 30 
#define CUDA_THREADS_PER_BLOCK 256	//this should be a multiple of 32 
#define MAX_FILLING 10
#define MAX_SITES 20

#define PI 3.141592653589793f

// calculated the momentum that each boson is at and populates the monomials array with this. Does so in ascending order.
__device__ void get_monomial_bosons_cuda(unsigned long Index, int Sites, int Bosons, int *Monomial) 
  {
    int mom = 1;
    int idx = 0;
    int pos = 0;
    while ( pos < (Sites + Bosons) ) {
      if ( (Index & (1ul << pos) ) > 0ul ) {
	  Monomial[idx++] = mom;
      } else {
	  mom += 1;
      }	  
      pos++;
    }  
  }

// calculated the momentum that each boson is at and populates the monomials array with this. Does so in ascending order and also return the normal.
__device__ void get_monomial_bosons_cuda(unsigned long Index, int Sites, int Bosons, int *Monomial, float *Normal) 
  {
    int mom = 1;
    int idx = 0;
    int pos = 0;
    *Normal = 1.0f;
    float num = 0;
    while ( pos < (Sites + Bosons) ) {
      if ( (Index & (1ul << pos) ) > 0ul ) {
	  Monomial[idx++] = mom;
	  num += 1.0f;
	  *Normal *= num;
      } else {
	  mom += 1;
	  //if ( num > 1.0 ) *normal *= num; 
	  num = 0.0f; 
      }	  
      pos++;
    }
    //if ( num > 1.0 ) *normal *= num; 
    *Normal = sqrtf(*Normal);
  }

// calculated the momentum that each boson is at and populates the monomials array with this. Does so in ascending order and also return the normal.
unsigned long convert_from_monomial_bosons(int Sites, int Bosons, int *Monomial) 
  {
    unsigned long config = 0;
    
    for ( int i = 0 ; i < Bosons ; i++ ) 
      {
	config += 1ul << (i + Monomial[Bosons-i-1]);
      }
    return config;     
  }


// calculated the momentum that each boson is at and populates the monomials array with this. Does so in ascending order and also return the normal.
__device__ unsigned long convert_from_monomial_bosons_cuda(int Sites, int Bosons, int *Monomial) 
  {
    unsigned long config = 0;
    
    for ( int i = 0 ; i < Bosons ; i++ ) 
      {
	config += 1ul << (i + Monomial[Bosons-i-1]);
      }
    return config;     
  }


// calculated the momentum that each boson is at and populates the monomials array with this. Does so in ascending order.
__device__ void get_monomial_fermions_cuda(unsigned long Index, int Sites, int Fermions, int *Monomial) 
  {
    int mom = 1;
    int idx = 0;
    int pos = 0;
    while ( pos < Sites ) {
      if ( (Index & (1ul << pos) ) > 0ul ) {
	  Monomial[idx++] = mom;
	  mom += 1; 
      } else {
	  mom += 1;
      }	  
      pos++;
    }  
  }

/*__device__ int permute_cuda(int *Monomial, int Len)
  {
    int k,l,tmp;
    
    k = Len - 2;
    while ((k > 0) && (Monomial[k] >= Monomial[k+1]) ) k--;
    
    if ( (k == 0) && (Monomial[0] >= Monomial[1]) ) return 0;
    
    l = Len - 1;
    while ( Monomial[l] <= Monomial[k] ) l--;
    
    tmp = Monomial[k];
    Monomial[k] = Monomial[l];
    Monomial[l] = tmp;
    
    l = Len - 1;
    k = k + 1;
    
    while ( k < l ) 
    {
      tmp = Monomial[k];
      Monomial[k] = Monomial[l];
      Monomial[l] = tmp;
      l--;
      k++;
    }    
    return 1;       
  }*/
  
__device__ int permute_cuda(int *Monomial, int Len)
  {
    int k,l,tmp;
    k = Len - 2;
    while ((k > 0) && (Monomial[k] >= Monomial[k+1]) ) k--;
    if ( (k == 0) && (Monomial[0] >= Monomial[1]) ) return 0;
    l = Len - 1;
    while ( Monomial[l] <= Monomial[k] ) l--;
    tmp = Monomial[k];
    Monomial[k] = Monomial[l];
    Monomial[l] = tmp;
    l = Len - 1;
    k = k + 1;
    while ( k < l ) 
    {
      tmp = Monomial[k];
      Monomial[k] = Monomial[l];
      Monomial[l] = tmp;
      l--;
      k++;
    }    
    return 1;       
  }  
  
__device__ float configuration_normal_cuda(unsigned long Config, int NumberSites, unsigned long Mask)
  {
    float normal = 1.0f;        
    for ( int i = 1 ; i < NumberSites ; i++ ) 
    {
      if ( Config == ( ((Config << i) | (Config >> (NumberSites - i))) & Mask ) ) 
	{
	  normal += 1.0f;
	}
    }
    return (1.0f/sqrtf(normal));
  }


__global__ void test_monomial_conversion(int *fmonomial, int NumberSites, int Filling, unsigned long Config)
  {
//    float stateNormal;
    get_monomial_fermions_cuda(Config, NumberSites, Filling, fmonomial);
    //while ( permute_cuda(fmonomial, Filling) ) {}     
  }

__global__ void calculate_FT_elements_alt(int VectorLength, float *VectorElements, unsigned long *BasisElements, int FTBasisDim, float *FTVectorElements, unsigned long *FTBasisElements, int NumberSites, int Filling, float *PhaseArrayR, int offset, float pre_norm) 
  {	    
    /*float l_PhaseArrayR[MAX_SITES];
    for ( int i = 0 ; i < NumberSites ; i++ ) 
      {
	l_PhaseArrayR[i] = PhaseArrayR[i];
      }*/
    int fMonomialPtr[MAX_FILLING];
    int fMonomialTmpPtr[MAX_FILLING];
    int oMonomialPtr[MAX_FILLING];
    __shared__ float shared_tmp_vals[CUDA_THREADS_PER_BLOCK];
    int fidx = blockIdx.x + offset * gridDim.x;
    if ( fidx < FTBasisDim ) 
      {	
	shared_tmp_vals[threadIdx.x] = 0.0f;
	float stateNormal;
	get_monomial_bosons_cuda(FTBasisElements[fidx], NumberSites, Filling, fMonomialPtr);
	get_monomial_bosons_cuda(FTBasisElements[fidx], NumberSites, Filling, fMonomialTmpPtr, &stateNormal);
	
	int oidx = threadIdx.x;
	while (oidx < VectorLength )
	  {
	    int phaseIdx = 0;
	    get_monomial_fermions_cuda(BasisElements[oidx], NumberSites, Filling, oMonomialPtr);	    
            for ( int i = 0; i < Filling; i++ )
              {
	        fMonomialPtr[i] = fMonomialTmpPtr[i];
		phaseIdx += oMonomialPtr[i] * fMonomialPtr[i];
              }
            float tmp_val2 = 0.0f;
	    int permuteFlg = 1, k, l, tmp;
	    while ( permuteFlg ) 
	      {
		tmp_val2 += PhaseArrayR[phaseIdx % NumberSites];
    
		k = Filling - 2;
		while ((k > 0) && (fMonomialPtr[k] >= fMonomialPtr[k+1]) ) k--;
		
		if ( (k == 0) && (fMonomialPtr[0] >= fMonomialPtr[1]) ) 
		  {
		    permuteFlg = 0;
		  } 
		else 
		  {
		    l = Filling - 1;
		    while ( fMonomialPtr[l] <= fMonomialPtr[k] ) l--;
		  
		    phaseIdx += (fMonomialPtr[l] - fMonomialPtr[k]) * oMonomialPtr[k] + (fMonomialPtr[k] - fMonomialPtr[l]) * oMonomialPtr[l];
		    tmp = fMonomialPtr[k];
		    fMonomialPtr[k] = fMonomialPtr[l];
		    fMonomialPtr[l] = tmp;
		    l = Filling - 1;
		    k = k + 1;
		    
		    while ( k < l ) 
		    {
		      phaseIdx += (fMonomialPtr[l] - fMonomialPtr[k]) * oMonomialPtr[k] + (fMonomialPtr[k] - fMonomialPtr[l]) * oMonomialPtr[l];
		      tmp = fMonomialPtr[k];
		      fMonomialPtr[k] = fMonomialPtr[l];
		      fMonomialPtr[l] = tmp;
		      l--;
		      k++;
		    }    
		    permuteFlg = 1;    
		  }
	      }
	    
	    shared_tmp_vals[threadIdx.x] += tmp_val2 * VectorElements[oidx] ;
	    oidx += blockDim.x;
	  }
    	  
	__syncthreads();
	int i = blockDim.x/2;
	while ( i != 0 ) 
	  {
	    if ( threadIdx.x < i )
	      {
	        shared_tmp_vals[threadIdx.x] += shared_tmp_vals[threadIdx.x + i];
	      }
	    __syncthreads();  
	    i /= 2;
	  }
	if ( threadIdx.x == 0 )  
	  {
	    FTVectorElements[fidx] = shared_tmp_vals[0] * pre_norm * stateNormal;
	  }
      }      
  }
  
__global__ void calculate_FT_elements(int VectorLength, float *VectorElements, unsigned long *BasisElements, int FTBasisDim, float *FTVectorElements, unsigned long *FTBasisElements, int NumberSites, int Filling, float *PhaseArrayR, int offset, float pre_norm) 
  {	            
    int fidx;     
    int fMonomialPtr[MAX_FILLING];
    int fMonomialTmpPtr[MAX_FILLING];
    int oMonomialPtr[MAX_FILLING];
    __shared__ float shared_tmp_vals[CUDA_THREADS_PER_BLOCK];
    float l_PhaseArrayR[MAX_SITES];
    
    for ( int i = 0 ; i < NumberSites ; i++ )
      {
        l_PhaseArrayR[i] = PhaseArrayR[i];
      }
    
    int  phaseIdx;    
    float stateNormal, tmp_val2 ;
        
    fidx = blockIdx.x + offset * gridDim.x;
    if ( fidx < FTBasisDim ) 
      {	
	shared_tmp_vals[threadIdx.x] = 0.0f;
	get_monomial_bosons_cuda(FTBasisElements[fidx], NumberSites, Filling, fMonomialPtr, &stateNormal);
	get_monomial_bosons_cuda(FTBasisElements[fidx], NumberSites, Filling, fMonomialTmpPtr, &stateNormal);
	
	int oidx = threadIdx.x;
	while (oidx < VectorLength )
	  {
	    get_monomial_fermions_cuda(BasisElements[oidx], NumberSites, Filling, oMonomialPtr);	    
            for ( int i = 0; i < Filling; i++ )
              {
	        fMonomialPtr[i] = fMonomialTmpPtr[i];
              }
            tmp_val2 = 0.0f;        
            do 
              {
	        phaseIdx = 0;
	        for ( int i = 0 ; i < Filling; i++ )
	          {
	            phaseIdx += oMonomialPtr[i] * fMonomialPtr[i];	            
	          }
	        tmp_val2 += l_PhaseArrayR[phaseIdx % NumberSites] ;
	      }
	    while ( permute_cuda(fMonomialPtr, Filling) );
	    shared_tmp_vals[threadIdx.x] += tmp_val2 * VectorElements[oidx] ;
	    oidx += blockDim.x;
	  }
    	  
	__syncthreads();
	int i = blockDim.x/2;
	while ( i != 0 ) 
	  {
	    if ( threadIdx.x < i )
	      {
	        shared_tmp_vals[threadIdx.x] += shared_tmp_vals[threadIdx.x + i];
	      }
	    __syncthreads();  
	    i /= 2;
	  }
	if ( threadIdx.x == 0 )  
	  {
	    FTVectorElements[fidx] = shared_tmp_vals[0] * pre_norm * stateNormal;
	  }
      }      
  }  
 
void fourier_transform_vector_cuda(int VectorLength, float *VectorElements, unsigned long *BasisElements, int FTBasisDim, float *FTVectorElements, unsigned long *FTBasisElements, int NumberSites, int Filling, int CurrentMomentum, int SpinMomentum, bool single, int CudaDevice, string OutputPrefix, int Offset)
  {        
    hipError_t cudaerr;
    //get device code
//     hipDeviceProp_t prop;
//     int dev;
//     hipGetDevice( &dev );    
//     memset( &prop, 0, sizeof( hipDeviceProp_t ) );
//     prop.multiProcessorCount = 16;    
//     hipChooseDevice( &dev, &prop );
//     cout << "ID of CUDA device with closest to 32 cores. " << dev << endl;    
    cudaerr = hipSetDevice( CudaDevice );
    cout << "Setting CUDA device to: " << CudaDevice << ": " << hipGetErrorString(cudaerr) << endl;  
    
    //prepare phase arrays
    float *phaseArrayR = new float[NumberSites+1];
    float *phaseArrayI = new float[NumberSites+1];
    phaseArrayR[0] = 1.0f;
    phaseArrayI[0] = 0.0f;
    for( int i = 1  ; i <= NumberSites ; i++ ) 
      {	    	    				
        float angle = ( PI * 2.0f * (float)i) / ((float)NumberSites);
	phaseArrayR[i] = (float)cos(angle);
	phaseArrayI[i] = (float)sin(angle);	
      }
      
    float norm, momentumPhase = 1.0f;    
    if ( SpinMomentum != -1 ) 
      {
	for ( int i = 1; i < NumberSites ; i++ )
	  {
	    momentumPhase += phaseArrayR[(i * (CurrentMomentum + SpinMomentum)) % NumberSites];
	  }
      }
    norm = sqrtf(momentumPhase) * sqrtf(1.0f/(float)powf((float)NumberSites,(float)Filling));  
                    
    //declare and allocate space.
    float  *d_FTVectorElements, *d_VectorElements, *d_phaseArrayR;
    unsigned long *d_FTBasisElements, *d_BasisElements;    
            
    cudaerr = hipMalloc( (void**) &d_phaseArrayR, (NumberSites + 1) * sizeof(float)); cout << "Allocating d_phaseArrayR: " << hipGetErrorString(cudaerr) << endl;   
    cudaerr = hipMalloc( (void**) &d_FTBasisElements, FTBasisDim * sizeof(unsigned long) ); cout << "Allocating d_FTBasisELements: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMalloc( (void**) &d_FTVectorElements, FTBasisDim * sizeof(float) );cout << "Allocating d_FTVectorElements: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMalloc( (void**) &d_BasisElements, VectorLength * sizeof(unsigned long) );cout << "Allocating d_BasisElements: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMalloc( (void**) &d_VectorElements, VectorLength * sizeof(float) );cout << "Allocating d_VectorElements: " << hipGetErrorString(cudaerr) << endl;
    
    cudaerr = hipMemcpy( d_phaseArrayR, phaseArrayR, (NumberSites + 1) * sizeof(float), hipMemcpyHostToDevice ); cout << "Copying phaseArrayR: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMemcpy( d_FTBasisElements, FTBasisElements, FTBasisDim * sizeof(unsigned long), hipMemcpyHostToDevice ); cout << "Copying FTBasisElements: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMemcpy( d_FTVectorElements, FTVectorElements, FTBasisDim * sizeof(float), hipMemcpyHostToDevice ); cout << "Copying FTVectorElements: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMemcpy( d_BasisElements, BasisElements, VectorLength * sizeof(unsigned long), hipMemcpyHostToDevice ); cout << "Copying BasisELements: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMemcpy( d_VectorElements, VectorElements, VectorLength * sizeof(float), hipMemcpyHostToDevice ); cout << "Copying VectorElements: " << hipGetErrorString(cudaerr) << endl;
	
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int maxOffset = ceil((float)FTBasisDim / (float)CUDA_BLOCKS);
    if ( single ) maxOffset = 1;
    for ( int offset = Offset ; offset < maxOffset ; offset++ ) 
      {
	cout << "Offset: " << offset << " out of " << maxOffset << endl; 
	hipEventRecord( start, 0 );
	calculate_FT_elements_alt<<<CUDA_BLOCKS, CUDA_THREADS_PER_BLOCK>>>(VectorLength, d_VectorElements, d_BasisElements, FTBasisDim, d_FTVectorElements, d_FTBasisElements, NumberSites, Filling, d_phaseArrayR, offset, norm);
	hipEventRecord( stop, 0 );
	cudaerr = hipGetLastError(); cout << "Error after kernel launch: " << hipGetErrorString(cudaerr) << endl;       
	hipDeviceSynchronize();
	cudaerr = hipGetLastError(); cout << "Error after kernel finish: " << hipGetErrorString(cudaerr) << endl;       
	hipEventSynchronize( stop );
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop);
	cout << "Time from CUDA timer: " << elapsedTime << endl;	
	cudaerr = hipMemcpy( FTVectorElements, d_FTVectorElements, FTBasisDim * sizeof(float), hipMemcpyDeviceToHost ); cout << "Copying back FTVectorELements: " << hipGetErrorString(cudaerr) << endl;       
	string outputFilename;
	stringstream ss("");
	ss << OutputPrefix << "_FT_M_" << CurrentMomentum << "_offset_" << offset << ".vec" ; 
	ofstream out(ss.str().c_str());
	if ( out.is_open() )
	  {
	    out.precision(14);
	    for ( int i = FTBasisDim - 1 ; i >= 0 ; i-- ) 
	      {
		out << scientific << FTVectorElements[i] << endl;
	      }
	    out.close();
	  }	  
      }        
    

    /*for ( int i = 0 ; i < VectorLength ; i++ ) 
      {	
	test_monomial_conversion<<<1,1>>>(d_omonomial, NumberSites, Filling, BasisElements[i]);	    	
	err = hipMemcpy( omonomial, d_omonomial, Filling * sizeof(int), hipMemcpyDeviceToHost ); cout << "Copying back fmonomial: " << err << endl;
	
	cout << BasisElements[i] << ": " ;
	for ( int i = 0 ; i < Filling ; i++ ) 
	  {
	    cout << omonomial[i] << " "  ;
	  }
	cout << endl;
      }*/
    hipEventDestroy( start );
    hipEventDestroy( stop );
      
        
    hipFree(d_phaseArrayR);
    hipFree(d_FTVectorElements);
    hipFree(d_VectorElements);
    hipFree(d_FTBasisElements);
    hipFree(d_BasisElements);
          
    delete [] phaseArrayR;
    delete [] phaseArrayI;
  }

/*__device__ int compare(int *MonomialA, int *MonomialB, int Filling)
  {
    for ( int i = 0 ; i < Filling ; i++ ) 
      {
	if (MonomialA[i] < MonomialB[i] ) 
	  {
	    return 0;
	  } 
	else if (MonomialA[i] > MonomialB[i] ) 
	  {
	    return 1;
	  }
      }
    return 1;
  }
  
  
__global__ void calculate_FT_element(float *VectorElements, unsigned long *BasisElements, int VectorLength, unsigned long FTBasisElement, int NumberSites, int Filling, int CurrentMomentum, int SpinMomentum, float *PhaseArrayR, float *BlockSums, int *Permutations) 
  {	                    
    if ( blockIdx.x < VectorLength ) 
      {
	int fMonomialCurrPtr[MAX_FILLING];
	int fMonomialNextPtr[MAX_FILLING];
	int oMonomialPtr[MAX_FILLING];
	__shared__ float shared_tmp_vals[CUDA_THREADS_PER_BLOCK];
	shared_tmp_vals[threadIdx.x] = 0.0f;    
	
	int  phaseIdx;    
	float stateNormal, norm, momentumPhase;
	unsigned long mask = (1ul << NumberSites) - 1ul;            
	    
	get_monomial_bosons_cuda(FTBasisElement, NumberSites, Filling, fMonomialCurrPtr, &stateNormal); // just for state normal really
	
	momentumPhase = 1.0f;    
	if ( SpinMomentum != -1 ) 
	  {
	    for ( int i = 1; i < NumberSites ; i++ )
	      {
		momentumPhase += PhaseArrayR[(i * (CurrentMomentum + SpinMomentum)) % NumberSites];
	      }
	  }
	norm = sqrtf(momentumPhase) * sqrtf(1.0f/(float)powf((float)NumberSites,(float)Filling));
			    
	int globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
	
	for ( int i = 0 ; i < Filling; i++ )
	  {	    
	    fMonomialCurrPtr[i] = Permutations[threadIdx.x*Filling + i];
	    if ( threadIdx.x < (blockDim.x -1) )
	      {	    
		fMonomialNextPtr[i] = Permutations[(threadIdx.x +1) * Filling + i];
	      }
	  }
		    
	int permuteFlg = 1;
	while ( permuteFlg && ((threadIdx.x == (blockDim.x -1 )) || (compare(fMonomialCurrPtr, fMonomialNextPtr, Filling) == 0)) )    
	  {
	    int oidx = blockIdx.x;    
	    while ( oidx < VectorLength ) 
	      {  
		get_monomial_fermions_cuda(BasisElements[oidx], NumberSites, Filling, oMonomialPtr);	    	    
		phaseIdx = 0;
		for ( int i = 0 ; i < Filling; i++ )
		  {    
		    phaseIdx += oMonomialPtr[i] * fMonomialCurrPtr[i];	            
		  }
		shared_tmp_vals[threadIdx.x] += PhaseArrayR[phaseIdx % NumberSites] * VectorElements[oidx] * configuration_normal_cuda(BasisElements[oidx], NumberSites, mask);
		oidx += gridDim.x;
	      }
	    permuteFlg = permute_cuda(fMonomialCurrPtr, Filling);
	  }		      
		    
	__syncthreads();
	int i = blockDim.x/2;
	while ( i != 0 ) 
	  {
	    if ( threadIdx.x < i )
	      {
		shared_tmp_vals[threadIdx.x] += shared_tmp_vals[threadIdx.x + i];
	      }
	    __syncthreads();  
	    i /= 2;
	  }
	__syncthreads();  
	if ( threadIdx.x == 0 )  
	  {
	    BlockSums[blockIdx.x] = shared_tmp_vals[0] * norm * stateNormal;
	  }
      }
  }
  
long get_rank_chunk_start(long N, int Rank, int Size){
	long  max_chunk, min_chunk, cut_off; 
	
	min_chunk = (long)floor((double)N / (double)Size); 
	max_chunk = (long)ceil((double)N / (double)Size);
	cut_off = N - (Size * min_chunk); 

	if ( Rank < cut_off ) {
		return Rank * max_chunk;
	} else {
		return (cut_off * max_chunk) + ((Rank - cut_off) * min_chunk);
	}
}  
  
void find_boundary_permutations(int *InitialSet, int Filling, int NumberSites, int NumberPartitions, int *Permutations)
  {
    //want to find how many permutations there can be. 
    int monomial[Filling];
    for ( int i = 0 ; i < Filling ; i++ )
      {
	monomial[i] = InitialSet[i];
      }
    long numberPermutations = 0; 
    
    while ( permute(monomial, Filling) ) 
      {
	numberPermutations++;
      }        
    
    //sort(monomial, monomial + Filling);    
    for ( int i = 0 ; i < Filling ; i++ )
      {
	monomial[i] = InitialSet[i];
      }
    long permutationCount = 0;
    int partitionIdx = 0;
    int lastIdx = 0;
    
    do 
      {	
	if ( permutationCount == get_rank_chunk_start(numberPermutations, partitionIdx, NumberPartitions) )
	  {
	      for ( int i = 0 ; i < Filling ; i++ ) 
		{
		  Permutations[partitionIdx * Filling + i] = monomial[i];		  
		}
	      lastIdx = partitionIdx;
	      partitionIdx++;
	  }
	permutationCount++;
      } 
    while ( permute(monomial, Filling ) && ( partitionIdx < NumberPartitions )  );    
    while ( partitionIdx < NumberPartitions ) 
      {
	for ( int i = 0 ; i < Filling ; i++ ) 
	  {
	    Permutations[partitionIdx * Filling + i] = Permutations[lastIdx * Filling + i];
	  }	
	partitionIdx++;
      }
  }


void fourier_transform_vector_cuda2(int VectorLength, float *VectorElements, unsigned long *BasisElements, int FTBasisDim, float *FTVectorElements, unsigned long *FTBasisElements, int NumberSites, int Filling, int CurrentMomentum, int SpinMomentum)
  {        
    //get device code
    hipDeviceProp_t prop;
    int dev;
    hipGetDevice( &dev );    
    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.multiProcessorCount = 16;    
    hipChooseDevice( &dev, &prop );
    cout << "ID of CUDA device with closest to 32 cores. " << dev << endl;
    hipSetDevice( dev );
    
    //prepare phase arrays
    float *phaseArrayR = new float[NumberSites+1];
    float *phaseArrayI = new float[NumberSites+1];
    phaseArrayR[0] = 1.0f;
    phaseArrayI[0] = 0.0f;
    for( int i = 1  ; i <= NumberSites ; i++ ) 
      {	    	    				
        float angle = ( PI * 2.0f * (float)i) / ((float)NumberSites);
	phaseArrayR[i] = (float)cos(angle);
	phaseArrayI[i] = (float)sin(angle);	
      }                           
        
    float *d_blockSums, *blockSums, *d_phaseArrayR, *d_VectorElements;
    unsigned long *d_BasisElements;
    int *d_permutations, *permutations;
    hipError_t cudaerr;    
    permutations = new int[CUDA_THREADS_PER_BLOCK * Filling];
    
    blockSums = new float[CUDA_BLOCKS];
    cudaerr = hipMalloc( (void**) &d_blockSums, CUDA_BLOCKS * sizeof(float)); cout << "Allocating d_blockSums: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMalloc( (void**) &d_permutations, CUDA_THREADS_PER_BLOCK * Filling * sizeof(int)); cout << "Allocating d_blockSums: " << hipGetErrorString(cudaerr) << endl;   
    cudaerr = hipMalloc( (void**) &d_phaseArrayR, (NumberSites + 1) * sizeof(float)); cout << "Allocating d_phaseArrayR: " << hipGetErrorString(cudaerr) << endl;       
    //cudaerr = hipMalloc( (void**) &d_number_permutations, CUDA_BLOCKS * CUDA_THREADS_PER_BLOCK * sizeof(int)); cout << "Allocating d_number_permutations: " << hipGetErrorString(cudaerr) << endl;   
    cudaerr = hipMalloc( (void**) &d_BasisElements, VectorLength * sizeof(unsigned long) );cout << "Allocating d_BasisElements: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMalloc( (void**) &d_VectorElements, VectorLength * sizeof(float) );cout << "Allocating d_VectorElements: " << hipGetErrorString(cudaerr) << endl;
       
    cudaerr = hipMemcpy( d_phaseArrayR, phaseArrayR, (NumberSites + 1) * sizeof(float), hipMemcpyHostToDevice ); cout << "Copying phaseArrayR: " << hipGetErrorString(cudaerr) << endl;    	    
    cudaerr = hipMemcpy( d_BasisElements, BasisElements, VectorLength * sizeof(unsigned long), hipMemcpyHostToDevice ); cout << "Copying BasisELements: " << hipGetErrorString(cudaerr) << endl;
    cudaerr = hipMemcpy( d_VectorElements, VectorElements, VectorLength * sizeof(float), hipMemcpyHostToDevice ); cout << "Copying VectorElements: " << hipGetErrorString(cudaerr) << endl;
    int monomial[Filling];    
    
    double start, finish, max, start2, finish2;
    
    max = 0.0;
    
    for ( int fidx = 0 ; fidx < FTBasisDim ; fidx++ )
    //for ( int fidx = 6 ; fidx < 7 ; fidx++ ) 
      {
	start2 = MPI_Wtime();
	get_monomial_bosons(FTBasisElements[fidx], NumberSites, Filling, monomial);
	// int count = 0;
// 	do {
// 	  cout << count++ << ": " << convert_from_monomial_bosons(NumberSites, Filling, monomial) << ": ";
// 	  for ( int i = 0 ; i < Filling ; i++ )
// 	    {
// 	      cout << monomial[i] << ", ";
// 	    }
// 	  cout << endl;
// 	} while ( permute(monomial, Filling) );
    
	FTVectorElements[fidx] = 0.0f;		
	find_boundary_permutations(monomial, Filling, NumberSites, CUDA_THREADS_PER_BLOCK, permutations); 	
	//int *number_permutations = new int[CUDA_BLOCKS * CUDA_THREADS_PER_BLOCK];	
	// for ( int i = 0 ; i < CUDA_BLOCKS * CUDA_THREADS_PER_BLOCK ; i++ ) 
// 	  {
// 	    cout << i << ": " ;
// 	    for ( int j = 0 ; j < Filling ; j++ )
// 	      {
// 		cout << permutations[i*Filling + j]  << " ";
// 	      }
// 	      cout << convert_from_monomial_bosons(NumberSites, Filling, &permutations[i*Filling]);
// 	      cout << endl;
// 	  }
	cudaerr = hipMemcpy( d_permutations, permutations, CUDA_THREADS_PER_BLOCK * Filling * sizeof(int), hipMemcpyHostToDevice); //cout << "Copying permutations: " << hipGetErrorString(cudaerr) << endl;	
	
	for ( int i = 0 ; i < CUDA_BLOCKS; i++ ) 
	  {
	    blockSums[i] = 0.0f;
	  }
	cudaerr = hipMemcpy( d_blockSums, blockSums, CUDA_BLOCKS * sizeof(float), hipMemcpyHostToDevice); //cout << "Copying blockSums: " << hipGetErrorString(cudaerr) << endl;  
	start = MPI_Wtime();
	calculate_FT_element<<<CUDA_BLOCKS, CUDA_THREADS_PER_BLOCK>>>(d_VectorElements, d_BasisElements, VectorLength, FTBasisElements[fidx], NumberSites, Filling, CurrentMomentum, SpinMomentum, d_phaseArrayR, d_blockSums, d_permutations);
	finish = MPI_Wtime();
	if ( (finish - start) > max) max = finish - start; 
		
	cudaerr = hipGetLastError(); //cout << "Error after kernel execution: " << hipGetErrorString(cudaerr) << endl;   	    
	//cudaerr = hipMemcpy( number_permutations, d_number_permutations, CUDA_BLOCKS * CUDA_THREADS_PER_BLOCK  * sizeof(int), hipMemcpyDeviceToHost ); //cout << "Copying back number_permutations: " << hipGetErrorString(cudaerr) << endl;
	cudaerr = hipMemcpy( blockSums, d_blockSums, CUDA_BLOCKS * sizeof(float), hipMemcpyDeviceToHost );// cout << "Copying back blockSums: " << hipGetErrorString(cudaerr) << endl;	    
	int total_permutations = 0;
	// for ( int idx = 0 ; idx < CUDA_BLOCKS ; idx++ )
// 	  {		
// 	    total_permutations = 0;
// 	    for ( int j = 0 ; j < CUDA_THREADS_PER_BLOCK ; j++ ) 
// 	      {
// 		  total_permutations += number_permutations[idx * CUDA_THREADS_PER_BLOCK + j]; 		  
// 	      }
// 	    cout << "Block: " << idx << ": " << total_permutations << endl;
// 	  }	    
// 	cout << "Fidx: " << fidx << ": " << FTVectorElements[fidx] << ", permutations: " << endl; 
	
	for ( int idx = 0 ; idx < CUDA_BLOCKS ; idx++ )
	  {
	    FTVectorElements[fidx] += blockSums[idx];
	  }
	finish2 = MPI_Wtime();
	cout << "Fidx: " << fidx << ": " << FTVectorElements[fidx] << ", time: " << finish2 - start2 << endl;  
      }        
    cout << "Max time taken for element was: " << max << endl;
        
    hipFree(d_phaseArrayR);
    hipFree(d_blockSums);
    hipFree(d_permutations);
    hipFree(d_VectorElements);    
    hipFree(d_BasisElements);
          
    delete [] phaseArrayR;
    delete [] phaseArrayI;
    delete [] blockSums;
    delete [] permutations;
  }*/


